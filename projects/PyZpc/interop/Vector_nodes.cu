#include "hip/hip_runtime.h"
#include "Vector.hpp"
#include "zensim/ZpcFunctional.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include <fmt/core.h>
#include <tuple>
#include <variant>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/zeno.h>

namespace zeno {
struct MakeZsVector : INode {
    void apply() override {
        // TODO
        auto input_size = get_input2<int>("size");
        auto input_memsrc = get_input2<std::string>("memsrc");
        auto intput_devid = get_input2<int>("dev_id");
        // auto input_virtual = get_input2<bool>("virtual");
        auto intput_elem_type = get_input2<std::string>("elem_type");

        zs::memsrc_e memsrc;
        if (input_memsrc == "host")
            memsrc = zs::memsrc_e::host;
        else if (input_memsrc == "device")
            memsrc = zs::memsrc_e::device;
        else
            memsrc = zs::memsrc_e::um;

#define MAKE_VECTOR_OBJ_T(T)                                                                   \
    if (intput_elem_type == #T) {                                                              \
        auto allocator = zs::get_memory_source(memsrc, static_cast<zs::ProcID>(intput_devid)); \
        vectorObj->set(zs::Vector<T, zs::ZSPmrAllocator<false>>{allocator, 0});                \
    }

        auto vectorObj = std::make_shared<ZsVectorObject>();
        MAKE_VECTOR_OBJ_T(int)
        MAKE_VECTOR_OBJ_T(float)
        MAKE_VECTOR_OBJ_T(double)
        std::visit([input_size](auto &vec) { vec.resize(input_size); }, vectorObj->value);

        set_output("ZsVector", std::move(vectorObj));
    }
};

//  memsrc, size, elem_type, dev_id, virtual
ZENDEFNODE(MakeZsVector, {
                             {{"int", "size", "0"},
                              {"enum host device um", "memsrc", "device"},
                              {"int", "dev_id", "0"},
                              //   {"bool", "virtual", "false"},
                              {"enum float int double", "elem_type", "float"}},
                             {"ZsVector"},
                             {},
                             {"PyZFX"},
                         });

struct ReduceZsVector : INode {
    void apply() override {
        auto vectorObj = get_input<ZsVectorObject>("ZsVector");
        auto opStr = get_input2<std::string>("op");
        auto &vector = vectorObj->value;

        float result;
        std::visit(
            [&result, &opStr](auto &vector) {
                auto pol = zs::cuda_exec();
                using vector_t = RM_CVREF_T(vector);
                using val_t = typename vector_t::value_type;
                zs::Vector<val_t> res{1, zs::memsrc_e::device, 0};
                if (opStr == "add")
                    zs::reduce(pol, std::begin(vector), std::end(vector), std::begin(res), static_cast<val_t>(0),
                               zs::plus<val_t>{});
                else if (opStr == "max")
                    zs::reduce(pol, std::begin(vector), std::end(vector), std::begin(res), zs::limits<val_t>::min(),
                               zs::getmax<val_t>{});
                else
                    zs::reduce(pol, std::begin(vector), std::end(vector), std::begin(res), zs::limits<val_t>::max(),
                               zs::getmin<val_t>{});
                result = static_cast<float>(res.getVal());
            },
            vector);
        set_output2("result", result);
    }
};

ZENDEFNODE(ReduceZsVector, {
                               {"ZsVector", {"enum add max min", "op", "add"}},
                               {"result"},
                               {},
                               {"PyZFX"},
                           });

struct CopyZsVectorTo : INode {
    void apply() override {
        auto vectorObj = get_input<ZsVectorObject>("ZsVector");
        auto prim = get_input<PrimitiveObject>("prim");
        auto attr = get_input2<std::string>("attr");
        auto &vector = vectorObj->value;

        float result;
        std::visit(
            [&prim, &attr](auto &vector) {
                using vector_t = RM_CVREF_T(vector);
                using val_t = typename vector_t::value_type;
                if constexpr (zs::is_same_v<val_t, float> || zs::is_same_v<val_t, int>) {
                    if (prim->size() != vector.size())
                        fmt::print("BEWARE! copy sizes mismatch!\n");

                    auto &dst = prim->attr<val_t>(attr);

                    std::memcpy(dst.data(), vector.data(), sizeof(val_t) * vector.size());
                }
            },
            vector);

        set_output2("prim", prim);
    }
};

ZENDEFNODE(CopyZsVectorTo, {
                               {"ZsVector", "prim", {"string", "attr", "clr"}},
                               {"prim"},
                               {},
                               {"PyZFX"},
                           });
} // namespace zeno